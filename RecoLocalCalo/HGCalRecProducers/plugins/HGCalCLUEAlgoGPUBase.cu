#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "RecoLocalCalo/HGCalRecProducers/plugins/HGCalCLUEAlgoGPUBase.h"

HGCalCLUEAlgoGPUBase::HGCalCLUEAlgoGPUBase(float pDc, float pKappa, float pEcut,
					   float pOutlierDeltaFactor,
					   const HGCCLUESoA& pCLUESoA,
					   uint32_t nhits)
  : mDc(pDc), mKappa(pKappa), mEcut(pEcut), mOutlierDeltaFactor(pOutlierDeltaFactor), mCLUESoA(pCLUESoA), mNHits(nhits)
{
  mPad = calculate_padding(mNHits);
}

HGCalCLUEAlgoGPUBase::HGCalCLUEAlgoGPUBase(const HGCCLUESoA& pCLUESoAHost,
					   const ConstHGCCLUESoA& pCLUESoADev,
					   uint32_t nhits)
  : mCLUESoAHost(pCLUESoAHost), mCLUESoADev(pCLUESoADev), mNHits(nhits)
{
  mPad = calculate_padding(mNHits);
}

HGCalCLUEAlgoGPUBase::~HGCalCLUEAlgoGPUBase() { free_device(); }
    
void HGCalCLUEAlgoGPUBase::free_device() {
  // algorithm internal variables
  hipFree(mDevHist);
  hipFree(mDevSeeds);
  hipFree(mDevFollowers);
}

void HGCalCLUEAlgoGPUBase::allocate_common_memory_blocks() {
  hipMalloc(&mDevHist, sizeof(LayerTilesGPU) * NLAYERS);
  hipMalloc(&mDevSeeds, sizeof(cms::cuda::VecArray<int,clue_gpu::maxNSeeds>) );
  hipMalloc(&mDevFollowers, sizeof(cms::cuda::VecArray<int,clue_gpu::maxNFollowers>)*mNHits);
}

void HGCalCLUEAlgoGPUBase::set_memory() {
  // condense into single memset??
  hipMemset(mCLUESoA.rho,           0x00, sizeof(float)*mPad);
  hipMemset(mCLUESoA.delta,         0x00, sizeof(float)*mPad);
  hipMemset(mCLUESoA.nearestHigher, 0x00, sizeof(int)*mPad);
  hipMemset(mCLUESoA.clusterIndex,  0x00, sizeof(int)*mPad);
  hipMemset(mCLUESoA.isSeed,        0x00, sizeof(int)*mPad);
  
  // algorithm internal variables
  hipMemset(mDevHist, 0x00, sizeof(LayerTilesGPU) * NLAYERS);
  hipMemset(mDevSeeds, 0x00, sizeof(GPU::VecArray<int,clue_gpu::maxNSeeds>));
  hipMemset(mDevFollowers, 0x00, sizeof(GPU::VecArray<int,clue_gpu::maxNFollowers>)*mNHits);
}

void HGCalCLUEAlgoGPUBase::copy_tohost(const hipStream_t& stream) {
  //the original standalone version transferred only the cluster index
  hipMemcpyAsync(mCLUESoAHost.rho, mCLUESoADev.rho, mPad*mCLUESoAHost.nbytes, hipMemcpyDeviceToHost);
}

uint32_t HGCalCLUEAlgoGPUBase::calculate_padding(uint32_t nhits) {
  //align to warp boundary (assumption: warpSize = 32)
  return ((nhits - 1) / 32 + 1) * 32;
}

float HGCalCLUEAlgoGPUBase::calculate_block_multiplicity(unsigned nelements, unsigned nthreads) {
  return ceil(nelements/static_cast<float>(nthreads));
}

cms::cuda::device::unique_ptr<std::byte[]>
HGCalCLUEAlgoGPUBase::allocate_soa_memory_block(uint32_t st, const hipStream_t &stream) {
  return cms::cuda::make_device_unique<std::byte[]>(mPad * st, stream);
}
