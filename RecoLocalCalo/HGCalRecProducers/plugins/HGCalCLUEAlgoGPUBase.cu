#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "RecoLocalCalo/HGCalRecProducers/plugins/HGCalCLUEAlgoGPUBase.h"

HGCalCLUEAlgoGPUBase::HGCalCLUEAlgoGPUBase(float pDc, float pKappa, float pEcut,
					   float pOutlierDeltaFactor,
					   const HGCCLUEHitsSoA& pCLUEHitsSoA,
					   const HGCCLUEClustersSoA& pCLUEClustersSoA)
  : mDc(pDc), mKappa(pKappa), mEcut(pEcut), mOutlierDeltaFactor(pOutlierDeltaFactor), mCLUEHitsSoA(pCLUEHitsSoA), mCLUEClustersSoA(pCLUEClustersSoA)
{
  mNHits = mCLUEHitsSoA.nhits;
  mNClusters = mCLUEClustersSoA.nclusters;
  
  mPadHits = calculate_padding( mNHits );
  mPadClusters = calculate_padding( mNClusters );

  hipMalloc(&mDevHist, sizeof(HeterogeneousHGCalLayerTiles) * NLAYERS);
  hipMalloc(&mDevSeeds, sizeof(cms::cuda::VecArray<int,clue_gpu::maxNSeeds>) );
  hipMalloc(&mDevFollowers, sizeof(cms::cuda::VecArray<int,clue_gpu::maxNFollowers>)*mNHits);

  was_memory_allocated = true;
}

HGCalCLUEAlgoGPUBase::HGCalCLUEAlgoGPUBase(unsigned nhits, unsigned nclusters,
					   const HGCCLUEHitsSoA& pCLUEHitsSoAHost, const ConstHGCCLUEHitsSoA& pCLUEHitsSoADev,
					   const HGCCLUEClustersSoA& pCLUEClustersSoAHost, const ConstHGCCLUEClustersSoA& pCLUEClustersSoADev)
  : mNHits(nhits), mNClusters(nclusters), mCLUEHitsSoAHost(pCLUEHitsSoAHost), mCLUEHitsSoADev(pCLUEHitsSoADev), mCLUEClustersSoAHost(pCLUEClustersSoAHost), mCLUEClustersSoADev(pCLUEClustersSoADev)
{
  mPadHits = calculate_padding( mNHits );
  mPadClusters = calculate_padding( mNClusters );

  was_memory_allocated = false;
}

HGCalCLUEAlgoGPUBase::~HGCalCLUEAlgoGPUBase() {
  if(was_memory_allocated)
    free_device();
}
    
void HGCalCLUEAlgoGPUBase::free_device() {
  // algorithm internal variables
  hipFree(mDevHist);
  hipFree(mDevSeeds);
  hipFree(mDevFollowers);
}

void HGCalCLUEAlgoGPUBase::set_memory() {
  // condense into single memset??
  hipMemset(mCLUEHitsSoA.rho,           0x00, sizeof(float)*mPadHits);
  hipMemset(mCLUEHitsSoA.delta,         0x00, sizeof(float)*mPadHits);
  hipMemset(mCLUEHitsSoA.nearestHigher, 0x00, sizeof(int32_t)*mPadHits);
  hipMemset(mCLUEHitsSoA.clusterIndex,  0x00, sizeof(int32_t)*mPadHits);
  hipMemset(mCLUEHitsSoA.id,            0x00, sizeof(uint32_t)*mPadHits);
  hipMemset(mCLUEHitsSoA.isSeed,        0x00, sizeof(bool)*mPadHits);

  hipMemset(mCLUEClustersSoA.energy,    0x00, sizeof(float)*mPadClusters);
  hipMemset(mCLUEClustersSoA.x,         0x00, sizeof(float)*mPadClusters);
  hipMemset(mCLUEClustersSoA.y,         0x00, sizeof(float)*mPadClusters);
  hipMemset(mCLUEClustersSoA.seedId,    0x02, sizeof(uint32_t)*mPadClusters);

  // algorithm internal variables
  hipMemset(mDevHist, 0x00, sizeof(HeterogeneousHGCalLayerTiles) * NLAYERS);
  hipMemset(mDevSeeds, 0x00, sizeof(GPU::VecArray<int,clue_gpu::maxNSeeds>));
  hipMemset(mDevFollowers, 0x00, sizeof(GPU::VecArray<int,clue_gpu::maxNFollowers>)*mNHits);  
}


void HGCalCLUEAlgoGPUBase::copy_tohost(const hipStream_t& stream) {
  //the original standalone version transferred only the cluster index
  hipMemcpyAsync(mCLUEHitsSoAHost.rho, mCLUEHitsSoADev.rho,
		  mPadHits*mCLUEHitsSoAHost.nbytes, hipMemcpyDeviceToHost, stream);

  hipMemcpyAsync(mCLUEClustersSoAHost.energy, mCLUEClustersSoADev.energy,
		  mPadClusters*mCLUEClustersSoAHost.nbytes, hipMemcpyDeviceToHost, stream);
}

uint32_t HGCalCLUEAlgoGPUBase::calculate_padding(uint32_t n) {
  //align to warp boundary (assumption: warpSize = 32)
  return ((n - 1) / 32 + 1) * 32;
}

float HGCalCLUEAlgoGPUBase::calculate_block_multiplicity(unsigned nelements, unsigned nthreads) {
  return ceil(nelements/static_cast<float>(nthreads));
}

cms::cuda::device::unique_ptr<std::byte[]>
HGCalCLUEAlgoGPUBase::allocate_soa_memory_block(uint32_t st, const hipStream_t &stream) {
  return cms::cuda::make_device_unique<std::byte[]>(mPadHits * st, stream);
}
