#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "RecoLocalCalo/HGCalRecProducers/plugins/HGCalCLUEAlgoGPUBase.h"

HGCalCLUEAlgoGPUBase::HGCalCLUEAlgoGPUBase(float pDc, float pKappa, float pEcut,
					   float pOutlierDeltaFactor,
					   const HGCCLUEHitsSoA& pCLUEHitsSoA,
					   const HGCCLUEHitsSoA& pCLUEHitsSoA,
					   uint32_t nhits, uint32_t nclusters)
  : mDc(pDc), mKappa(pKappa), mEcut(pEcut), mOutlierDeltaFactor(pOutlierDeltaFactor), mCLUEHitsSoA(pCLUEHitsSoA), mCLUEClustersSoA(pCLUEClustersSoA),
    mNHits(nhits), mNClusters(nclusters)
{
  mPadHits = calculate_padding(mNHits);
  mPadClusters = calculate_padding(mNClusters);

  hipMalloc(&mDevHist, sizeof(LayerTilesGPU) * NLAYERS);
  hipMalloc(&mDevSeeds, sizeof(cms::cuda::VecArray<int,clue_gpu::maxNSeeds>) );
  hipMalloc(&mDevFollowers, sizeof(cms::cuda::VecArray<int,clue_gpu::maxNFollowers>)*mNHits);

  was_memory_allocated = true;
}

HGCalCLUEAlgoGPUBase::HGCalCLUEAlgoGPUBase(const HGCCLUEHitsSoA& pCLUEHitsSoAHost, const ConstHGCCLUEHitsSoA& pCLUEHitsSoADev,
					   const HGCCLUEClustersSoA& pCLUEClustersSoAHost, const ConstHGCCLUEClustersSoA& pCLUEClustersSoADev,
					   uint32_t nhits)
  : mCLUEHitsSoAHost(pCLUESoAHost), mCLUEHitsSoADev(pCLUESoADev), mCLUEClustersSoAHost(pCLUESoAHost), mCLUEClustersSoADev(pCLUESoADev), mNHits(nhits)
{
  mPadHits = calculate_padding(mNHits);
  mPadClusters = calculate_padding(mNClusters);

  was_memory_allocated = false;
}

HGCalCLUEAlgoGPUBase::~HGCalCLUEAlgoGPUBase() {
  if(was_memory_allocated)
    free_device();
}
    
void HGCalCLUEAlgoGPUBase::free_device() {
  // algorithm internal variables
  hipFree(mDevHist);
  hipFree(mDevSeeds);
  hipFree(mDevFollowers);
}

void HGCalCLUEAlgoGPUBase::set_memory() {
  // condense into single memset??
  hipMemset(mCLUEHitsSoA.rho,           0x00, sizeof(float)*mPadHits);
  hipMemset(mCLUEHitsSoA.delta,         0x00, sizeof(float)*mPadHits);
  hipMemset(mCLUEHitsSoA.nearestHigher, 0x00, sizeof(int)*mPadHits);
  hipMemset(mCLUEHitsSoA.clusterIndex,  0x00, sizeof(int)*mPadHits);
  hipMemset(mCLUEHitsSoA.isSeed,        0x00, sizeof(int)*mPadHits);
  
  // algorithm internal variables
  hipMemset(mDevHist, 0x00, sizeof(LayerTilesGPU) * NLAYERS);
  hipMemset(mDevSeeds, 0x00, sizeof(GPU::VecArray<int,clue_gpu::maxNSeeds>));
  hipMemset(mDevFollowers, 0x00, sizeof(GPU::VecArray<int,clue_gpu::maxNFollowers>)*mNHits);
}

void HGCalCLUEAlgoGPUBase::copy_tohost(const hipStream_t& stream) {
  //the original standalone version transferred only the cluster index
  hipMemcpyAsync(mCLUEHitsSoAHost.rho, mCLUEHitsSoADev.rho,
		  mPadHits*mCLUEHitsSoAHost.nbytes, hipMemcpyDeviceToHost, stream);

  hipMemcpyAsync(mCLUEClustersSoAHost.energy, mCLUEClustersSoADev.energy,
		  mPadClusters*mCLUEClustersSoAHost.nbytes, hipMemcpyDeviceToHost, stream);
}

uint32_t HGCalCLUEAlgoGPUBase::calculate_padding(uint32_t n) {
  //align to warp boundary (assumption: warpSize = 32)
  return ((n - 1) / 32 + 1) * 32;
}

float HGCalCLUEAlgoGPUBase::calculate_block_multiplicity(unsigned nelements, unsigned nthreads) {
  return ceil(nelements/static_cast<float>(nthreads));
}

cms::cuda::device::unique_ptr<std::byte[]>
HGCalCLUEAlgoGPUBase::allocate_soa_memory_block(uint32_t st, const hipStream_t &stream) {
  return cms::cuda::make_device_unique<std::byte[]>(mPadHits * st, stream);
}
